
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <limits.h>

#define TILE_WIDTH 16

void fill_matrix(double *mat, unsigned numRows, unsigned numCols) 
{ 
    for(unsigned i=0; i < numRows; i++) 
        for(unsigned j=0; j < numCols; j++)    
            mat[i*numCols + j] = i*2.1f + j*3.2f;    
}

void print_matrix_to_file(double *mat, unsigned numRows, unsigned numCols) 
{ 
    const char *fname = "assignment2_out"; 
    FILE *f = fopen(fname, "w"); 
 
    for(unsigned i=0; i < numRows; i++) 
    { 
        for(unsigned j=0; j < numCols; j++) 
            fprintf(f,"%4.4f ", mat[i*numCols + j]); 
        fprintf(f,"\n"); 
    } 
    fclose(f); 
}

__global__ void matmul(double* d_a, double* d_b, double* d_c, int n)
{
    // fastest varying index is y

    __shared__ double ds_a[TILE_WIDTH][TILE_WIDTH];
    __shared__ double ds_b[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;
    
    double res = 0;
    for(int p=0; p<n/TILE_WIDTH; p++)
    {
        ds_a[ty][tx] = d_a[row*n + p*TILE_WIDTH + tx];
        ds_b[ty][tx] = d_b[(p*TILE_WIDTH + ty)*n + col];
        __syncthreads();

        for(int i=0; i<TILE_WIDTH; i++)
            res += ds_a[ty][i]*ds_b[i][tx];
        __syncthreads();
    }

    d_c[row*n + col] = res;
}

int main()
{	
	srand(time(NULL));
    int n = 8192;
	int size = n*n;

    float milliseconds;
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    double *h_a = (double*)malloc(sizeof(double)*size); fill_matrix(h_a, n, n);
    double *h_b = (double*)malloc(sizeof(double)*size); fill_matrix(h_b, n, n);
    double *h_c = (double*)malloc(sizeof(double)*size);

    double *d_a; hipMalloc((void**)&d_a, sizeof(double)*size);
    double *d_b; hipMalloc((void**)&d_b, sizeof(double)*size);
    double *d_c; hipMalloc((void**)&d_c, sizeof(double)*size); 

    hipMemcpy(d_a, h_a, sizeof(double)*size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(double)*size, hipMemcpyHostToDevice);

    hipEventRecord(start);
    matmul<<<dim3(n/TILE_WIDTH,n/TILE_WIDTH),dim3(TILE_WIDTH,TILE_WIDTH)>>>(d_a, d_b, d_c, n);
    hipEventRecord(stop);
	hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("elapsed time = %f ms\n", milliseconds);

    hipMemcpy(h_c, d_c, sizeof(double)*size, hipMemcpyDeviceToHost);
    print_matrix_to_file(h_c, n, n);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

	return 0;
}