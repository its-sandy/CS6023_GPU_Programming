
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <limits.h>

void fill_matrix(double *mat, unsigned numRows, unsigned numCols) 
{ 
    for(unsigned i=0; i < numRows; i++) 
        for(unsigned j=0; j < numCols; j++)    
            mat[i*numCols + j] = i*2.1f + j*3.2f;    
}

void print_matrix_to_file(double *mat, unsigned numRows, unsigned numCols) 
{ 
    const char *fname = "assignment2_out"; 
    FILE *f = fopen(fname, "w"); 
 
    for(unsigned i=0; i < numRows; i++) 
    { 
        for(unsigned j=0; j < numCols; j++) 
            fprintf(f,"%4.4f ", mat[i*numCols + j]); 
        fprintf(f,"\n"); 
    } 
    fclose(f); 
}

__global__ void matrix_multiplication(double* d_a, double* d_b, double* d_c, int n)
{
    // fastest varying index is y
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

	double res = 0;
    for(int i=0; i<n; i++)
        res += d_a[row*n + i]*d_b[i*n + col];
    d_c[row*n + col] = res;
}

int main()
{	
	srand(time(NULL));
	int size, n, i;
    n = 2048; // 8192 = 1<<13
    size = n*n;

    float milliseconds;
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    double *h_a = (double*)malloc(sizeof(double)*size); fill_matrix(h_a, n, n);
    double *h_b = (double*)malloc(sizeof(double)*size); fill_matrix(h_b, n, n);
    double *h_c = (double*)malloc(sizeof(double)*size);

    double *d_a; hipMalloc((void**)&d_a, sizeof(double)*size);
    double *d_b; hipMalloc((void**)&d_b, sizeof(double)*size);
    double *d_c; hipMalloc((void**)&d_c, sizeof(double)*size); 

    hipMemcpy(d_a, h_a, sizeof(double)*size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(double)*size, hipMemcpyHostToDevice);

    /* we try
    (1<<11, 1<<11), (1<<2, 1<<2)
    (1<<11, 1<<10), (1<<2, 1<<3)
    (1<<10, 1<<10), (1<<3, 1<<3)
    (1<<10, 1<<9), (1<<3, 1<<4)
    (1<<9, 1<<9), (1<<4, 1<<4)
    (1<<9, 1<<8), (1<<4, 1<<5)
    (1<<8, 1<<8), (1<<5, 1<<5)

    (1<<9, 1<<7), (1<<4, 1<<6)
    */

    printf("Run Times for n = %d\n",n);
    for(i=4; i<=10 /*change according to half of max threads per block*/; i++)
	{
        hipEventRecord(start);
        matrix_multiplication<<<dim3(n/(1<<(i/2)),n/(1<<((i+1)/2))),dim3(1<<(i/2),1<<((i+1)/2))>>>(d_a, d_b, d_c, n);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Grid Structure: (%d, %d); Block Structure: (%d, %d); Time (ms): %f\n", n/(1<<(i/2)), n/(1<<((i+1)/2)), 1<<(i/2), 1<<((i+1)/2), milliseconds);
	}

    hipEventRecord(start);
    matrix_multiplication<<<dim3(n/(1<<4),n/(1<<6)),dim3(1<<4,1<<6)>>>(d_a, d_b, d_c, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Grid Structure: (%d, %d); Block Structure: (%d, %d); Time (ms): %f\n", n/(1<<4), n/(1<<6), 1<<4, 1<<6, milliseconds);


    hipMemcpy(h_c, d_c, sizeof(double)*size, hipMemcpyDeviceToHost);
    // print_matrix_to_file(h_c, n, n);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

	return 0;
}