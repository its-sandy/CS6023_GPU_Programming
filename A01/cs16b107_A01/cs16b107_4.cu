
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <limits.h>

__global__ void vecAdd(int* a, int* b, int* c)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	c[i] = a[i] + b[i];
}

__host__ void generateRandomVector(int* vec, int size)
{
	int i;

	for(i=0; i<size; i++)
		vec[i] = rand() % (INT_MAX/2); // so that addition does not cause overflow
}

__host__ void printResult(int* a, int* b, int* c, int size)
{
	int i;
	for(i=0; i<size; i++)
		printf("%d %d %d\n", a[i], b[i], c[i]);
}

__host__ float computeTime(int* a, int* b, int* c, int blocks, int threadsPerBlock, int iterations)
{
	int i;
	float totalTime = 0, milliseconds;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for(i=0; i<iterations; i++)
	{
		hipEventRecord(start);
		vecAdd<<<blocks,threadsPerBlock>>>(a, b, c);
		hipEventRecord(stop);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&milliseconds, start, stop);
		totalTime += milliseconds;
	}
	return totalTime/iterations;
}

int main()
{	
	int size = (1<<15), i;

	srand(time(NULL));
	int *h_a = (int*)malloc(sizeof(int)*size); generateRandomVector(h_a, size);
	int *h_b = (int*)malloc(sizeof(int)*size); generateRandomVector(h_b, size);
	int *h_c = (int*)malloc(sizeof(int)*size);

	int *d_a; hipMalloc((void**)&d_a, sizeof(int)*size);
	int *d_b; hipMalloc((void**)&d_b, sizeof(int)*size);
	int *d_c; hipMalloc((void**)&d_c, sizeof(int)*size); 

	hipMemcpy(d_a, h_a, sizeof(int)*size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(int)*size, hipMemcpyHostToDevice);

	for(i=0; i<=10 /*change according to max threads per block*/; i++)
	{
		printf("Blocks: (1<<%d), threadsPerBlock: (1<<%d), Average Time (ms): %f\n", 15-i, i, computeTime(d_a, d_b, d_c, (1<<(15-i)), (1<<i), 100));
	}

	// vecAdd<<<(1<<7),(1<<8)>>>(d_a, d_b, d_c);

	hipMemcpy(h_c, d_c, sizeof(int)*size, hipMemcpyDeviceToHost);

	// printResult(h_a, h_b, h_c, size);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	free(h_a);
	free(h_b);
	free(h_c);

	return 0;
}